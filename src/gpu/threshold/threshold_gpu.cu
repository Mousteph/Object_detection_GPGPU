#include "hip/hip_runtime.h"
#include "threshold_gpu.hpp"

#include <cassert>
#include <iostream>

#define cudaCheckError()                                                       \
    {                                                                          \
        hipError_t e = hipGetLastError();                                    \
        if (e != hipSuccess)                                                  \
        {                                                                      \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__,           \
                   hipGetErrorString(e));                                     \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

__global__ void apply_first_threshold(unsigned char *buffer, size_t rows, size_t cols, size_t pitch, int threshold) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col >= cols || row >= rows || buffer[col + row * pitch] >= threshold)
        return;

    buffer[col + row * pitch] = 0; 
}

__global__ void apply_bin_threshold(unsigned char *buffer, size_t rows, size_t cols, size_t pitch, int threshold) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col >= cols || row >= rows)
        return;

    buffer[col + row * pitch] = 255 * (buffer[col + row * pitch] >= threshold); 
}

void threshold(unsigned char *buffer, size_t rows, size_t cols, size_t pitch) {
    unsigned char otsu_thresh = otsu_threshold(buffer, rows, cols, pitch);
    unsigned char otsu_thresh2 = otsu_thresh * 2.5;

    dim3 threads(32,32);
    dim3 blocks(std::ceil(float(cols) / float(threads.x)),
                std::ceil(float(rows) / float(threads.y)));

    apply_first_threshold<<<blocks, threads>>>(buffer, rows, cols, pitch, otsu_thresh - 10);
    cudaCheckError();
    hipDeviceSynchronize();
    
    apply_bin_threshold<<<blocks, threads>>>(buffer, rows, cols, pitch, otsu_thresh2);
    cudaCheckError();
    hipDeviceSynchronize();
}

