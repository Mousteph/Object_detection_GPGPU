#include "hip/hip_runtime.h"
#include "detect_obj_gpu.hpp"
#include <cassert>
#include <iostream>
#include "helpers_gpu.hpp"


__global__ void propagate2(unsigned char *buffer_base, unsigned int *buffer_bin,
                          size_t rows, size_t cols, size_t pitch, size_t pitch_bin, bool *has_change) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col >= cols || row >= rows || *eltPtr<unsigned char>(buffer_base, col, row, pitch) == 0)
        return;

    unsigned int* b_init = eltPtr<unsigned int>(buffer_bin, col, row, pitch_bin);
    unsigned int current = *b_init;


    if (col + 1 < cols && *eltPtr<unsigned int>(buffer_bin, col + 1, row, pitch_bin) != 0)
    {
        unsigned int val = *eltPtr<unsigned int>(buffer_bin, col + 1, row, pitch_bin);
        current = current == 0 ? val : min(current, val);
    }
    if (row + 1 < rows && *eltPtr<unsigned int>(buffer_bin, col, row + 1, pitch_bin) != 0)
    {
        unsigned int val = *eltPtr<unsigned int>(buffer_bin, col, row + 1, pitch_bin);
        current = current == 0 ? val : min(current, val);
    }
    if (col - 1 < cols && *eltPtr<unsigned int>(buffer_bin, col - 1, row, pitch_bin) != 0)
    {
        unsigned int val = *eltPtr<unsigned int>(buffer_bin, col - 1, row, pitch_bin);
        current = current == 0 ? val : min(current, val);
    }
    if (row - 1 < rows && *eltPtr<unsigned int>(buffer_bin, col, row - 1, pitch_bin) != 0)
    {
        unsigned int val = *eltPtr<unsigned int>(buffer_bin, col, row - 1, pitch_bin);
        current = current == 0 ? val : min(current, val);
    }

    if (*b_init != current)
    {
        *has_change = true;
        *b_init = current;
    }
}

__global__ void mask_label(unsigned int *buffer_bin, unsigned char *labelled, size_t rows, size_t cols, size_t pitch_bin) {

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col >= cols || row >= rows)
        return;

    unsigned int* bin = eltPtr<unsigned int>(buffer_bin, col, row, pitch_bin);
    if (*bin == 0)
        return;

    unsigned int v = *bin;
    if (labelled[v] == (unsigned char) 0) {
        labelled[v] = (unsigned char) 1;
    }
}

__global__ void continous_labels(unsigned char *labels, size_t rows, size_t cols, int *val) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col >= cols || row >= rows || labels[col + row * cols] == 0)
        return;
    
    int old = atomicAdd(val, 1);
    labels[col + row * cols] = old;
}

__global__ void relabelled(unsigned char *buffer, unsigned int *buffer_bin, unsigned char *labelled,
                           size_t rows, size_t cols, size_t pitch, size_t pitch_bin) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col >= cols || row >= rows)
        return;

    unsigned int* bin = eltPtr<unsigned int>(buffer_bin, col, row, pitch_bin);
    unsigned char* buf = eltPtr<unsigned char>(buffer, col, row, pitch);

    if (*bin == 0)
        *buf = 0;
    else {
        *buf = labelled[*bin];
    }
}

__global__ void set_value(bool *has_change, bool val) {
	*has_change = val;
}

__global__ void apply_bin_threshold(unsigned int *buffer_bin, unsigned char *buffer_base, size_t rows, size_t cols,
                                     size_t pitch, size_t pitch_bin, int threshold) {
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col >= cols || row >= rows)
        return;

    unsigned int val = col + row * cols + 1;
    unsigned int *b_bin = (unsigned int *)((char*)buffer_bin + row * pitch_bin + col * sizeof(unsigned int));

    if (buffer_base[col + row * pitch] >= threshold)
        *b_bin = val;
    else
        *b_bin = 0;
}

bool get_has_change(bool *d_has_change) {
    bool h_has_change;
    hipMemcpy(&h_has_change, d_has_change, sizeof(bool), hipMemcpyDeviceToHost);
    return h_has_change;
}

int connexe_components(unsigned char *buffer_base, size_t rows, size_t cols, size_t pitch, unsigned char threshold, int thx, int thy) {
    dim3 threads(thx, thy);
    dim3 blocks(std::ceil(float(cols) / float(threads.x)), std::ceil(float(rows) / float(threads.y)));
    
    size_t pitch_bin;
    unsigned int *buffer_bin = malloc2Dcuda<unsigned int>(rows, cols, &pitch_bin);

    apply_bin_threshold<<<blocks, threads>>>(buffer_bin, buffer_base, rows, cols, pitch, pitch_bin, threshold);
    hipDeviceSynchronize();
    cudaCheckError();

    bool *d_has_change = mallocCpy<bool>(false, sizeof(bool));
    bool h_has_change = true;

    while (h_has_change) {
	set_value<<<1, 1>>>(d_has_change, false);
        for (int i = 0; i < 5; i++) {
            propagate2<<<blocks, threads>>>(buffer_base, buffer_bin, rows, cols, pitch, pitch_bin, d_has_change);
        }
        hipDeviceSynchronize();
        cudaCheckError();
	h_has_change = get_has_change(d_has_change);
    }

    int h_nb_compo = 1;
    int *d_nb_compo = mallocCpy<int>(1, sizeof(int));

    unsigned char *labels = malloc1Dcuda<unsigned char>(sizeof(unsigned char) * rows * cols);
    hipMemset(labels, 0, rows * cols * sizeof(unsigned char));

    mask_label<<<blocks, threads>>>(buffer_bin, labels, rows, cols, pitch_bin);
    hipDeviceSynchronize();
    cudaCheckError();
    
    continous_labels<<<blocks, threads>>>(labels, rows, cols, d_nb_compo);
    hipDeviceSynchronize();
    cudaCheckError();

    relabelled<<<blocks, threads>>>(buffer_base, buffer_bin, labels, rows, cols, pitch, pitch_bin);
    hipDeviceSynchronize();
    cudaCheckError();
    
    hipMemcpy(&h_nb_compo, d_nb_compo, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_nb_compo);
    hipFree(labels);
    hipFree(buffer_bin);

    return h_nb_compo - 1;
}
