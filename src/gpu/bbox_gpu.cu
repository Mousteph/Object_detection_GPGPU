#include "hip/hip_runtime.h"
#include "detect_obj_gpu.hpp"

__global__ void bbox(unsigned char *buffer, int *maxh, int *minh, int *maxw, int *minw, size_t rows, size_t cols, size_t pitch){

   int col = blockDim.x * blockIdx.x + threadIdx.x;
   int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col >= cols || row >= rows) {
        return;
    }

    unsigned char myval = buffer[col + row * pitch];
  
    if (myval > 0) {
      atomicMax(maxw + myval - 1, col);
      atomicMin(minw + myval - 1, col);
      atomicMax(maxh + myval - 1, row);
      atomicMin(minh + myval - 1, row);
    }
}

struct Bbox** get_bbox(unsigned char *buffer, size_t rows, size_t cols, size_t pitch, int nb_components) {
    int max = nb_components;
    int *maxw, *maxh, *minw, *minh, *d_maxw, *d_maxh, *d_minw, *d_minh;
   
    maxw = new int[max];
    maxh = new int[max];
    minw = new int[max];
    minh = new int[max];
  
    hipMalloc(&d_maxw, max * sizeof(int));
    hipMalloc(&d_maxh, max * sizeof(int));
    hipMalloc(&d_minw, max * sizeof(int));
    hipMalloc(&d_minh, max * sizeof(int));
  
    for (int i = 0; i < max; i++) {
        maxw[i] = 0;
        maxh[i] = 0;
        minw[i] = cols;
        minh[i] = rows;
    }

    hipMemcpy(d_maxw, maxw, max * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_maxh, maxh, max * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_minw, minw, max * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_minh, minh, max * sizeof(int), hipMemcpyHostToDevice);

    dim3 threads(32, 32);
    dim3 blocks(std::ceil(float(cols) / float(threads.x)),
                std::ceil(float(rows) / float(threads.y)));

    bbox<<<blocks, threads>>>(buffer, d_maxh, d_minh, d_maxw, d_minw, rows, cols, pitch);
  
    hipMemcpy(maxw, d_maxw, max * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(maxh, d_maxh, max * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(minw, d_minw, max * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(minh, d_minh, max * sizeof(int), hipMemcpyDeviceToHost);
    
    struct Bbox** bboxes = (struct Bbox**) std::malloc(nb_components * sizeof(struct Bbox *));

    for (int i = 0; i < max; i++) {
        struct Bbox* bbox = (struct Bbox*) std::malloc(sizeof(struct Bbox));
        bbox->x = minw[i];
        bbox->y = minh[i];
        bbox->height = maxh[i] - minh[i];
        bbox->width = maxw[i] - minw[i];

        bboxes[i] = bbox;
    }

    hipFree(d_maxh);
    hipFree(d_maxw);
    hipFree(d_minh);
    hipFree(d_minw);

    return bboxes;
}
