#include "helpers_gpu.hpp"


int getProperties(int device, hipDeviceProp_t *deviceProp)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (device >= deviceCount)
        return -1;

    hipGetDeviceProperties(deviceProp, device);
    if (deviceProp->major == 9999 && deviceProp->minor == 9999)
    {
        std::cerr << "No CUDA GPU has been detected" << std::endl;
        return -1;
    }
    return 0;
}
