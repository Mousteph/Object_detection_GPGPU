#include "detect_obj.hpp"
#include <spdlog/spdlog.h>
#include <cassert>
#include <iostream>

[[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
  hipError_t err = hipGetLastError();
  spdlog::error("{} ({}, line: {})", msg, fname, line);
  spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)


void detect_gpu(unsigned char *buffer_ref, unsigned char *buffer_obj, int width, int height, int channels) {
    return;
}

