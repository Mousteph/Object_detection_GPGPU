#include "detect.hpp"
#include <spdlog/spdlog.h>
#include <cassert>

[[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
  hipError_t err = hipGetLastError();
  spdlog::error("{} ({}, line: {})", msg, fname, line);
  spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)


void detect_cpu(char* buffer_start, char *buffer_obj, int width, int height, std::ptrdiff_t stride) {
    return;
}

void detect_gpu(char* buffer_start, char *buffer_obj, int width, int height, std::ptrdiff_t stride) {
    return;
}
